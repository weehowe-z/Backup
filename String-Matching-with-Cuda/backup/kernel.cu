#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <vector>
#include <string>
#include <fstream>
#include <iostream>
#include <stdlib.h>
#include <cstring>
using namespace std;

#define NUM_THREADS 1000
#define NUM_BLOCKS 10000
#define NB 4
char txt[40000001];
char pat[4001];



__device__ void computeLPSArray(const char *pat, int M, int *lps)
{
    int len = 0;  // length of the previous longest prefix suffix
    int i;

    lps[0] = 0; // lps[0] is always 0
    i = 1;

    // the loop calculates lps[i] for i = 1 to M-1
    while (i < M)
    {
       if (pat[i] == pat[len])
       {
         len++;
         lps[i] = len;
         i++;
       }
       else // (pat[i] != pat[len])
       {
         if (len != 0)
         {
           // This is tricky. Consider the example
           // AAACAAAA and i = 7.
           len = lps[len-1];

           // Also, note that we do not increment i here
         }
         else // if (len == 0)
         {
           lps[i] = 0;
           i++;
         }
       }
    }
}


__device__ void KMPSearch(int*c, int* lps, const char *pat, const char *txt, int sizepat, int sizetxt, int bid)
{
    int M = sizepat;
    int N = sizetxt;
    int j  = 0;  // index for pat[]

    // Preprocess the pattern (calculate lps[] array)
	// moved to the kernel function
    
	//printf("lps finished\n");
	int x=0;
    int i = 0;  // index for txt[]
    while (i < N)
    {
      if (pat[j] == txt[i])
      {
        j++;
        i++;
      }

      if (j == M)
      {
          cout<<("Found pattern at index %d \n", i-j + bid*NB);
		  c[bid* NB + x] = i-j;
		  x++;
		  j = lps[j-1];
      }
      // mismatch after j matches
      else if (i < N && pat[j] != txt[i])
      {
        // Do not match lps[0..lps[j-1]] characters,
        // they will match anyway
        if (j != 0)
         j = lps[j-1];
        else
         i = i+1;
      }
    }
    free(lps); // to avoid memory leak
	//printf("finished\n");
}


void addWithCuda(int *c, const char *a, const char *b, unsigned int sizec, int sizea, int sizeb);

// __global__ void kmpKernel(int *c, int *lps, const char *a, const char *b, int sizea, int sizeb)
// {
// 	//int tid = threadIdx.x;
// 	int bid = blockIdx.x;
// 	//int idx = bid * blockDim.x + tid;
// 	if ( bid*NB <sizea)
// 	{
// 		computeLPSArray(b, sizeb, lps);
// 		KMPSearch(c, lps, b,a+bid*NB, sizeb, NB + sizeb - 1, bid);

// 	}
// }


__global__ void kmpKernel(int *c, int *lps, const char *txt, const char *pat, int sizetxt, int sizepat)
{
  int tid = threadIdx.x;
  int bid = blockIdx.x;
  int idx = bid * blockDim.x + tid;
  //printf("Found pattern at index %d \n", bid);
  if (idx*NB <sizetxt-sizepat)
  {
    computeLPSArray(pat, sizepat, lps);
    KMPSearch(c, lps, pat, txt + idx*NB, sizepat, NB + sizepat - 1, idx);

  }
  if (idx*NB >sizetxt-sizepat && (idx + 1)*NB<sizetxt-sizepat)
  {
    computeLPSArray(pat, sizepat, lps);
    KMPSearch(c, lps, pat, txt + idx*NB, sizepat, sizetxt - idx*NB, idx);
  }
}

int main()
{
	// ifstream fin("input.txt");
	// ofstream fout("output.txt");
	// //char *txt = new char[4000] ;
	// //char *txt = "AABAACAADAABAAABAA";
	// //char *pat = new char[1000];

	// int *rs = new int[4001];
	// for (int i=0;i<1000;++i) rs[i] = -1;

	// fin.getline(txt,40000001,'\n');
 //    fin.getline(pat,4001,'\n');

	// //cout << txt <<"\n"<< pat <<"\n";
	// const int sizec = 4001;
	
	// int len_txt = strlen(txt);
	// int len_pat = strlen(pat);
	
	// cout<<"haha\n";
	// addWithCuda(rs, txt, pat, sizec, len_txt, len_pat);
	// cout<<"haha\n";
 
  ifstream fin("input.txt");
  ofstream fout("out.txt");
  char *txt = new char[40000000];
  //char *txt = "AABAACAADAABAAABAA";
  char *pat = new char[4000];
  int *rs = new int[40000000];
  for (int i = 0; i<40000000; ++i) rs[i] = -1;

  fin.getline(txt, 40000000, '\n');
  fin.getline(pat, 4000, '\n');

  //cout << txt << "\n" << pat << "\n";
  const int sizec = 40000000;

  int len_txt = strlen(txt);
  int len_pat = strlen(pat);


  addWithCuda(rs, txt, pat, sizec, len_txt, len_pat);


  return 0;
}

// Helper function for using CUDA to add vectors in parallel.
void addWithCuda(int *c, const char *a, const char *b, unsigned int sizec, int sizea, int sizeb)
{
    char *dev_a = 0;
    char *dev_b = 0;
    int *dev_c = 0;
	int *dev_lps = 0;
   

    // Allocate GPU buffers for three vectors (two input, one output)    .
    hipMalloc((void**)&dev_c, sizec * sizeof(int));
 
	hipMalloc((void**)&dev_lps, sizeb * sizeof(int));


    hipMalloc((void**)&dev_a, sizea * sizeof(char));
   

    hipMalloc((void**)&dev_b, sizeb * sizeof(char));

    // Copy input vectors from host memory to GPU buffers.
    hipMemcpy(dev_a, a, sizea * sizeof(char), hipMemcpyHostToDevice);
    hipMemcpy(dev_b, b, sizeb * sizeof(char), hipMemcpyHostToDevice);

    // Launch a kernel on the GPU with one thread for each element.
    kmpKernel<<<NUM_BLOCKS, 1>>>(dev_c, dev_lps, dev_a, dev_b, sizea, sizeb);

    
    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    hipDeviceSynchronize();

    // Copy output vector from GPU buffer to host memory.
    hipMemcpy(c, dev_c, sizec * sizeof(int), hipMemcpyDeviceToHost);

    hipFree(dev_c);
    hipFree(dev_a);
    hipFree(dev_b);
	hipFree(dev_lps);

}
