
#include <hip/hip_runtime.h>
#include <iostream>
#include <fstream>
#include <string>
#include <time.h>
using namespace std;


void preKMP(string pattern, int next[]);
void KMPSearch(string target, string pattern);
void BFSearch(string target, string pattern);

int main(int argc, char const *argv[])
{
	ifstream in("input.txt");
	if (!in) {
		cerr << "error, cannot find input.txt\n";
		return -1;
	}

	string target,pattern;
	in >> target;
	in >> pattern;
	// BFSearch(target,pattern);
	KMPSearch(target,pattern);



	ofstream output("output2.txt");
	return 0;
}

void BFSearch(string target, string pattern)
{
	clock_t begin, end;
	double time_spent;

	cout<<"--------------------\n";
	cout<<"Alogrithm:\tBrute Force\n\n";
	begin = clock();

	int target_len = target.length();
	int pattern_len = pattern.length();
	int i = 0;

	while (i <= target_len - pattern_len){
		int j = 0;
		while (j < pattern_len){
			if ((target[i+j]) != pattern[j]) break;
			++j;
		}
		if (j == pattern_len) {
			cout<<"Match @ position "<< i << endl;
		}
		++i;
	}

	end = clock();
	time_spent = (double)(end - begin)* 1000 / CLOCKS_PER_SEC;
	cout<<"\ntime spent:\t"<<time_spent<<"ms"<<endl;
	cout<<"--------------------\n";
}

void preKMP(string pattern,int next[])
{
	int pattern_len = pattern.length();
	int k; //  longest suffix
    next[0] = 0;
    for (int i = 1; i < pattern_len; ++i)
    {
		while(k > 0 && pattern[i] != pattern[k]) k = next[k-1];
		if (pattern[i] == pattern[k]) ++k;
		next[i] = k;	
    }
}


void KMPSearch(string target, string pattern){
	clock_t begin, end;
	double time_spent;

	cout<<"--------------------\n";
	cout<<"Alogrithm:\tKMP\n\n";
	begin = clock();

	int target_len = target.length();
	int pattern_len = pattern.length();
	int i = 0;
	int j = 0;
	int *next = new int[pattern_len];

	preKMP(pattern,next);

	while (i <= target_len - pattern_len){
		while (j < pattern_len){
			if ((target[i+j]) != pattern[j]) break;
			++j;
		}
		if (j == 0){
			++i;
		}
		else if  (j == pattern_len) {
			j = 0;
			++i;
		}
		else {
			i += j - next[j-1];
			j = next[j-1]; 
		}
	}


	delete []next;

	end = clock();
	time_spent = (double)(end - begin)* 1000 / CLOCKS_PER_SEC;
	cout<<"\ntime spent:\t"<<time_spent<<"ms"<<endl;
	cout<<"--------------------\n";

}