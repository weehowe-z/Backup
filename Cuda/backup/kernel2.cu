#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <vector>
#include <string>
#include <fstream>
#include <iostream>
#include <stdlib.h>
#include <cstring>
using namespace std;


#define NUM_THREADS 1000
#define NUM_BLOCKS 10000
#define NB 4


// void preKMP2(string pattern, unsigned int next[]);
// void KMPSearch2(string target, string pattern);


__device__ void preKMP(const char *pat, int M, int *lps)
{
	int len = 0;  // length of the previous longest prefix suffix
	int i;

	lps[0] = 0; // lps[0] is always 0
	i = 1;

	// the loop calculates lps[i] for i = 1 to M-1
	while (i < M)
	{
		if (pat[i] == pat[len])
		{
			len++;
			lps[i] = len;
			i++;
		}
		else // (pat[i] != pat[len])
		{
			if (len != 0)
			{
				// This is tricky. Consider the example
				// AAACAAAA and i = 7.
				len = lps[len - 1];

				// Also, note that we do not increment i here
			}
			else // if (len == 0)
			{
				lps[i] = 0;
				i++;
			}
		}
	}
}
__device__ void KMPSearch(int*c, int* lps, const char *pat, const char *txt, int size_pattern, int size_target, int bid)
{
	int M = size_pattern;
	int N = size_target;
	int j = 0;  // index for pat[]

	// Preprocess the pattern (calculate lps[] array)
	// moved to the kernel function

	int i = 0;  // index for txt[]
	while (i < N)
	{
		if (pat[j] == txt[i])
		{
			j++;
			i++;
		}

		if (j == M)
		{
			printf("Found pattern at index %d \n", i - j + bid*NB);
			c[bid* NB +i - j] =1 ;
			
			j = lps[j - 1];
		}
		// mismatch after j matches
		else if (i < N && pat[j] != txt[i])
		{
			// Do not match lps[0..lps[j-1]] characters,
			// they will match anyway
			if (j != 0)
				j = lps[j - 1];
			else
				i = i + 1;
		}
	}
	free(lps);
}


void preCuda(int *c, const char *a, const char *b, unsigned int sizec, int sizea, int sizeb);

__global__ void kmpKernel(int *c, int *lps, const char *txt, const char *pat, int size_target, int size_pattern)
{
	int tid = threadIdx.x;
	int bid = blockIdx.x;
	int idx = bid * blockDim.x + tid;
	//printf("Found pattern at index %d \n", bid);
	if (idx*NB <size_target-size_pattern)
	{
		preKMP(pat, size_pattern, lps);
		KMPSearch(c, lps, pat, txt + idx*NB, size_pattern, NB + size_pattern - 1, idx);

	}
	if (idx*NB > size_target-size_pattern && (idx + 1) * NB < size_target-size_pattern)
	{
		preKMP(pat, size_pattern, lps);
		KMPSearch(c, lps, pat, txt + idx*NB, size_pattern, size_target - idx*NB, idx);
	}
}

int main()
{
	ifstream fin("input2.txt");

	// ifstream fin("input2.txt");
	ofstream fout("out.txt");
	// fout = fopen("out.txt","w");
	char *target = new char[40000000];
	char *pattern = new char[4000];
	int *rs = new int[40000000];
	for (int i = 0; i<40000000; ++i) rs[i] = -1;

	fin.getline(target, 40000000, '\n');
	fin.getline(pattern, 4000, '\n');

	//cout << target << "\n" << pattern << "\n";
	const int sizec = 40000000;

	int size_target = strlen(target);
	int size_pattern = strlen(pattern);

	preCuda(rs, target, pattern, sizec, size_target, size_pattern);

	for (int i = 0; i<40000000; ++i)
	{
		if (rs[i]==1	)
		fout << i <<  "\n";
	}

	delete []target;
	delete []pattern;

	return 0;
}

// Helper function for using CUDA to add vectors in parallel.
void preCuda(int *c, const char *a, const char *b, unsigned int sizec, int sizea, int sizeb)
{
	char *dev_a = 0;
	char *dev_b = 0;
	int *dev_c = 0;
	int *dev_lps = 0;

	// Choose which GPU to run on, change this on a multi-GPU system.
	// string target(a),pattern(b);
	// KMPSearch2(target,pattern);
	// return;
	// Allocate GPU buffers for three vectors (two input, one output)    .
	hipMalloc((void**)&dev_c, sizec * sizeof(int));
	hipMalloc((void**)&dev_lps, sizeb * sizeof(int));
	hipMalloc((void**)&dev_a, sizea * sizeof(char));
	hipMalloc((void**)&dev_b, sizeb * sizeof(char));

	// Copy input vectors from host memory to GPU buffers.
	hipMemcpy(dev_a, a, sizea * sizeof(char), hipMemcpyHostToDevice);


	hipMemcpy(dev_b, b, sizeb * sizeof(char), hipMemcpyHostToDevice);


	// Launch a kernel on the GPU with one thread for each element.
	kmpKernel << <NUM_BLOCKS, NUM_THREADS >> >(dev_c, dev_lps, dev_a, dev_b, sizea, sizeb);
	hipDeviceSynchronize();

	hipMemcpy(c, dev_c, sizec * sizeof(int), hipMemcpyDeviceToHost);


	hipFree(dev_c);
	hipFree(dev_a);
	hipFree(dev_b);
	hipFree(dev_lps);

}


void preKMP2(string pattern,unsigned int next[])
{
	unsigned int pattern_len = pattern.length();
	unsigned int k; //  longest suffix
    next[0] = 0;
    for (int i = 1; i < pattern_len; ++i)
    {
		while(k > 0 && pattern[i] != pattern[k]) k = next[k-1];
		if (pattern[i] == pattern[k]) ++k;
		next[i] = k;	
    }
}


void KMPSearch2(string target, string pattern){
	clock_t begin, end;
	double time_spent;

	cout<<"--------------------\n";
	cout<<"Alogrithm:\tKMP\n\n";
	begin = clock();

	unsigned int target_len = target.length();
	unsigned int pattern_len = pattern.length();
	unsigned int i = 0;
	unsigned int j = 0;
	unsigned int *next = new unsigned int[pattern_len];

	preKMP2(pattern,next);

	while (i <= target_len - pattern_len){
		// cout<<"OUT target["<<i<<"]\n";
		// cout<<"OUT pattern["<<j<<"]\n";
		while (j < pattern_len){
			// cout<<"inner "<<j<<endl;
			if ((target[i+j]) != pattern[j]) break;
			++j;
		}
		// cout<<"current j "<<j<<endl;
		if (j == 0){
			++i;
		}
		else if  (j == pattern_len) {
			cout<<"Match @ position "<< i << endl;
			j = 0;
			++i;
		}
		else {
			i += j - next[j-1];
			j = next[j-1]; 
			// cout<<"afedfdfd "<<j<<endl;
		}
	}


	delete []next;

	end = clock();
	time_spent = (double)(end - begin)* 1000 / CLOCKS_PER_SEC;
	cout<<"\ntime spent:\t"<<time_spent<<"ms"<<endl;
	cout<<"--------------------\n";

}