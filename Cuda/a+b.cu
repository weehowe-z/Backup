
#include <hip/hip_runtime.h>
#include <iostream>
#include <time.h>
using namespace std;

//Calculate on the kernel
__global__ void add(int* a, int* b, int* c)
{
	*c = *a + *b;
}


int main(int argc, char const *argv[])
{
	// clock_t begin, end;
	clock_t gpu_begin, gpu_end;
	// double time_spent;
	double gpu_time_spent;



	//host copy
	int a,b,c;
	a = 10;
	b = 20;
	c = 0;

	//device copy 
	int *d_a,*d_b,*d_c;

	int size = sizeof(int);

	cout<<"Begin to allocate GPU space\n";
	gpu_begin = clock();

	//allocate GPU space
	hipMalloc((void **)&d_a, size);
	hipMalloc((void **)&d_b, size);
	hipMalloc((void **)&d_c, size);

	//copy input to device
	hipMemcpy(d_a, &a, size, hipMemcpyHostToDevice);
	hipMemcpy(d_b, &b, size, hipMemcpyHostToDevice);

	add<<<1,1>>>(d_a, d_b, d_c);

	hipMemcpy(&c, d_c, size, hipMemcpyDeviceToHost);

	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);

	gpu_end = clock();
	gpu_time_spent = (double)(gpu_end - gpu_begin) / CLOCKS_PER_SEC;





	//kernel
	cout<<"c is "<<c<<endl;
	cout<<"time_spent "<<gpu_time_spent<<"s"<<endl;

	return 0;
}