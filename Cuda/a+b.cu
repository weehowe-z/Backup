
#include <hip/hip_runtime.h>
#include <iostream>
#include <time.h>
#include <stdlib.h>
using namespace std;

//Calculate on the kernel
__global__ void add(int* a, int* b, int* c)
{
	int i = threadIdx.x;
	*c = *a + *b;
}


int main(int argc, char const *argv[])
{
	int blocks = 1, threads= 1;
	if(argc == 3){
		blocks = atoi(argv[1]);
		threads = atoi(argv[2]);
	}
	cout<<"--------------------\n";
	cout<<"Blocks: " << blocks <<endl;
	cout<<"Threads: " << threads <<endl;
	cout<<"----------"<<endl;


	// clock_t begin, end;
	clock_t gpu_begin, gpu_end;
	// double time_spent;
	double gpu_time_spent;


	//host copy
	int a,b,c;
	a = 10;
	b = 20;
	c = 0;

	//device copy 
	int *d_a,*d_b,*d_c;

	int size = sizeof(int);

	gpu_begin = clock();

	//allocate GPU space
	hipMalloc((void **)&d_a, size);
	hipMalloc((void **)&d_b, size);
	hipMalloc((void **)&d_c, size);

	//copy input to device
	hipMemcpy(d_a, &a, size, hipMemcpyHostToDevice);
	hipMemcpy(d_b, &b, size, hipMemcpyHostToDevice);

	add<<<blocks,threads>>>(d_a, d_b, d_c);

	hipMemcpy(&c, d_c, size, hipMemcpyDeviceToHost);

	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);

	gpu_end = clock();
	gpu_time_spent = (double)(gpu_end - gpu_begin) / CLOCKS_PER_SEC;





	//kernel
	cout<<"c is "<<c<<endl;
	cout<<"time_spent "<<gpu_time_spent<<"s"<<endl;
	cout<<"--------------------\n\n";

	return 0;
}