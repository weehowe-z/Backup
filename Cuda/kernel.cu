#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <vector>
#include <string>
#include <fstream>
#include <iostream>
#include <stdlib.h>
#include <cstring>
using namespace std;

#define NUM_THREADS 1000
#define NUM_BLOCKS 10000
#define NB 4
char txt[40000001];
char pat[4001];
__device__ void computeLPSArray(const char *pat, int M, int *lps)
{
    int len = 0;  // length of the previous longest prefix suffix
    int i;

    lps[0] = 0; // lps[0] is always 0
    i = 1;

    // the loop calculates lps[i] for i = 1 to M-1
    while (i < M)
    {
       if (pat[i] == pat[len])
       {
         len++;
         lps[i] = len;
         i++;
       }
       else // (pat[i] != pat[len])
       {
         if (len != 0)
         {
           // This is tricky. Consider the example
           // AAACAAAA and i = 7.
           len = lps[len-1];

           // Also, note that we do not increment i here
         }
         else // if (len == 0)
         {
           lps[i] = 0;
           i++;
         }
       }
    }
}
__device__ void KMPSearch(int*c, int* lps, const char *pat, const char *txt, int sizepat, int sizetxt, int bid)
{
    int M = sizepat;
    int N = sizetxt;
    int j  = 0;  // index for pat[]

    // Preprocess the pattern (calculate lps[] array)
	// moved to the kernel function
    
	//printf("lps finished\n");
	int x=0;
    int i = 0;  // index for txt[]
    while (i < N)
    {
      if (pat[j] == txt[i])
      {
        j++;
        i++;
      }

      if (j == M)
      {
          printf("Found pattern at index %d \n", i-j + bid*NB);
		  c[bid* NB + x] = i-j;
		  x++;
		  j = lps[j-1];
      }
      // mismatch after j matches
      else if (i < N && pat[j] != txt[i])
      {
        // Do not match lps[0..lps[j-1]] characters,
        // they will match anyway
        if (j != 0)
         j = lps[j-1];
        else
         i = i+1;
      }
    }
    free(lps); // to avoid memory leak
	//printf("finished\n");
}


hipError_t addWithCuda(int *c, const char *a, const char *b, unsigned int sizec, int sizea, int sizeb);

__global__ void kmpKernel(int *c, int *lps, const char *a, const char *b, int sizea, int sizeb)
{
	//int tid = threadIdx.x;
	int bid = blockIdx.x;
	//int idx = bid * blockDim.x + tid;
	if ( bid*NB <sizea)
	{
		computeLPSArray(b, sizeb, lps);
		KMPSearch(c, lps, b,a+bid*NB, sizeb, NB + sizeb - 1, bid);

	}
}

int main()
{
	ifstream fin("G://input.txt");
	ofstream fout("G://out.txt");
	//char *txt = new char[4000] ;
	//char *txt = "AABAACAADAABAAABAA";
	//char *pat = new char[1000];

	int *rs = new int[4001];
	for (int i=0;i<1000;++i) rs[i] = -1;

	fin.getline(txt,40000001,'\n');
    fin.getline(pat,4001,'\n');

	//cout << txt <<"\n"<< pat <<"\n";
	const int sizec = 4001;
	
	int len_txt = strlen(txt);
	int len_pat = strlen(pat);
	

	hipError_t cudaStatus = addWithCuda(rs, txt, pat, sizec, len_txt, len_pat);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addWithCuda failed!");
        return 1;
    }
	
	int i;
	for (i=0;i<4001;++i)
	{
		fout << i <<"\t" <<  rs[i] << "\n";
	}
    
	cudaStatus = hipDeviceReset();
    //if (cudastatus != cudasuccess) {
    //    fprintf(stderr, "cudadevicereset failed!");
    //    return 1;
    //}

    return 0;
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t addWithCuda(int *c, const char *a, const char *b, unsigned int sizec, int sizea, int sizeb)
{
    char *dev_a = 0;
    char *dev_b = 0;
    int *dev_c = 0;
	int *dev_lps = 0;
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&dev_c, sizec * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }
	cudaStatus = hipMalloc((void**)&dev_lps, sizeb * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_a, sizea * sizeof(char));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_b, sizeb * sizeof(char));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_a, a, sizea * sizeof(char), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_b, b, sizeb * sizeof(char), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    // Launch a kernel on the GPU with one thread for each element.
    kmpKernel<<<NUM_BLOCKS, 1>>>(dev_c, dev_lps, dev_a, dev_b, sizea, sizeb);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    
    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(c, dev_c, sizec * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(dev_c);
    hipFree(dev_a);
    hipFree(dev_b);
	hipFree(dev_lps);
    
    return cudaStatus;
}
