
#include <hip/hip_runtime.h>
#include <iostream>
#include <time.h>
#include <stdlib.h>
#define N 8
using namespace std;

void random_ints(int* a, int num)
{
	for (int i = 0; i < num; ++i){
		a[i] = rand() % 20;
		cout<<a[i]<<"\t";
	}
	cout<<endl<<endl;
}


//Calculate on the kernel
__global__ void add(int* a, int* b, int* c)
{
	c[blockIdx.x] = a[blockIdx.x] + b[blockIdx.x];
}

int main(int argc, char const *argv[])
{
	int blocks = 1, threads= 1;
	if(argc == 3){
		blocks = atoi(argv[1]); 
		threads = atoi(argv[2]);
	}
	cout<<"--------------------\n";
	cout<<"Blocks: " << blocks <<endl;
	cout<<"Threads: " << threads <<endl;
	cout<<"----------"<<endl;

	srand(time(NULL));

	// clock_t begin, end;
	clock_t gpu_begin, gpu_end;
	// double time_spent;
	double gpu_time_spent;


	//host copy
	int *a,*b,*c;

	//device copy 
	int *d_a,*d_b,*d_c;

	int size = N * sizeof(int);

	a = (int *)malloc(size);
	b = (int *)malloc(size);
	c = (int *)malloc(size);

	for (int i=0; i<N; ++i){
		c[i] = 0;
	}
	random_ints(a, N);
	random_ints(b, N);


	//allocate GPU space
	hipMalloc((void **)&d_a, size);
	hipMalloc((void **)&d_b, size);
	hipMalloc((void **)&d_c, size);


	//copy input to device
	hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
	hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);
	hipMemcpy(d_c, c, size, hipMemcpyHostToDevice);
	gpu_begin = clock();

	add<<<blocks,threads>>>(d_a, d_b, d_c);

	gpu_end = clock();
	gpu_time_spent = (double)(gpu_end - gpu_begin)* 1000 / CLOCKS_PER_SEC;



	hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);

	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);

	for (int i=0; i<N; ++i){
		cout<< c[i]<<"\t";
	}
	cout<<endl<<endl<<endl;

	//kernel
	// cout<<"c is "<<c<<endl;
	cout<<"time_spent "<<gpu_time_spent<<"ms"<<endl;
	cout<<"--------------------\n\n";

	free(a);
	free(b);
	free(c);

	return 0;
}