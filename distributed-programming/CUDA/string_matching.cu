
#include <hip/hip_runtime.h>
#include <iostream>
#include <fstream>
// #include "time.h"
#define NUM_OF_THREADS 1024;

using namespace std;

void preKMP(char* pattern, int f[])
{
    int m = strlen(pattern), k;
    f[0] = -1;
    for (int i = 1; i < m; i++)
    {
        k = f[i - 1];
        while (k >= 0)
        {
            if (pattern[k] == pattern[i - 1])
                break;
            else
                k = f[k];
        }
        f[i] = k + 1;
    }
}
 
__global__ void KMP(char* pattern, char* target,int f[],int c[],int n, int m)
{
    int index = blockIdx.x*blockDim.x + threadIdx.x;
    int i = n * index;
    int j = n * (index + 2)-1;

    if(i>m)
        return;
    if(j>m)
        j=m;

    int k = 0;        
    while (i < j)
    {
        if (k == -1)
        {
            i++;
            k = 0;
        }
        else if (target[i] == pattern[k])
        {
            i++;
            k++;
            if (k == n)
            {
                c[i - n] = i-n;
                i = i - k + 1;
            }
        }
        else
            k = f[k];
    }
    return;
}
 
int main(int argc, char* argv[])
{
    ifstream fin;
    ofstream fout;

    if (argc != 3){
        cout<<"Plz specify input file and output file!\n";
        return -1;
    }

    fin.open(argv[1]);

    if (!fin){
        cout<<"Input file dose not exist!\n";
        return -1;       
    } 

    fout.open(argv[2]);


    cout<<"\n--------------------\n";
    cout<<"Input:\t"<<argv[1]<<endl;
    cout<<"Algorithm:\tKMP\n";


    const int L = 40000000;
    const int S = 40000000;
    int M = NUM_OF_THREADS;

    int cSize = 4;
    char *tar,*pat;
    tar = (char*)malloc(L*cSize);
    pat = (char*)malloc(S*cSize);
    
    //device copy
    char *d_tar, *d_pat;

    fin>>tar>>pat;

    int m = strlen(tar);
    int n = strlen(pat);

    cout<<"Target length:\t"<<m<<endl;
    cout<<"Pattern length:\t"<<n<<endl;
    cout<<"----\n";
    int *f;//next table
    int *c;//place table

    f = new int[m];
    c = new int[m];

    int *d_f;
    int *d_c;
    for(int i = 0;i<m; i++)
    {
        c[i] = -1;
    }     
    
    preKMP(pat, f);

    cout<<"Copy data to GPU\n";

    clock_t copy_start,copy_end,other_start,other_end;
    double copy_time_spent,total_time_spent;


    copy_start = clock();

    hipMalloc((void **)&d_tar, m*cSize);
    hipMalloc((void **)&d_pat, n*cSize);
    hipMalloc((void **)&d_f, m*cSize);
    hipMalloc((void **)&d_c, m*cSize);

    hipMemcpy(d_tar, tar, m*cSize, hipMemcpyHostToDevice);
    hipMemcpy(d_pat, pat, n*cSize, hipMemcpyHostToDevice);
    hipMemcpy(d_f, f, m*cSize, hipMemcpyHostToDevice);
    hipMemcpy(d_c, c, m*cSize, hipMemcpyHostToDevice);

    copy_end = clock();
    copy_time_spent = (double)(copy_end - copy_start)* 1000 / CLOCKS_PER_SEC;
    cout<<"Time spent:\t"<<copy_time_spent<<"ms"<<endl;
    cout<<"----\n";
    other_start = clock();

    // if(n>10000000)
    //     M = 128;

    float time_elapsed=0;
    hipEvent_t start,stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start,0);
    KMP<<<(m/n+M)/M,M>>>(d_pat, d_tar ,d_f, d_c, n, m);
    hipEventRecord(stop,0); 
 
    hipEventSynchronize(start);    //Waits for an event to complete.
    hipEventSynchronize(stop);    //Waits for an event to complete.Record之前的任务
    hipEventElapsedTime(&time_elapsed,start,stop);    //计算时间差

    cout<<"String Matching\n";
    printf("Time spent:\t%.2fms\n", time_elapsed);  
    cout<<"----\n";

    hipMemcpy(c, d_c, m*cSize, hipMemcpyDeviceToHost);

    for(int i = 0;i<m; i++)
    { 
        if(c[i]!=-1)
        {
            fout<<c[i]<<'\n';
        }
    }

    hipFree(d_tar); 
    hipFree(d_pat);
    hipFree(d_f);
    hipFree(d_c);
    
    delete []f;
    delete []c;



    other_end = clock();
    total_time_spent = (double)(copy_end - copy_start + other_end - other_start)* 1000 / CLOCKS_PER_SEC;
    cout<<"Total time:\t"<<total_time_spent<<"ms"<<endl;
    cout<<"--------------------\n\n";
    



    return 0;
}